#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUBLAS(call)                                                  \
{                                                                           \
    const hipblasStatus_t status = (call);                                    \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                    \
        std::cerr << "cuBLAS error occurred: " << status << std::endl; \
        std::exit(EXIT_FAILURE);                                            \
    }                                                                       \
}

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

int main() {
    const int N = 1024;
    const int K = 1024;
    const int M = 1024;

    // Allocate host memory for input matrices
    float* hostA = new float[N * K];
    float* hostB = new float[K * M];
    float* hostC = new float[N * M];

    // Initialize input matrices
    for (int i = 0; i < N * K; ++i) {
        hostA[i] = static_cast<float>(i);
    }

    for (int i = 0; i < K * M; ++i) {
        hostB[i] = static_cast<float>(i);
    }

    // Allocate device memory
    float* deviceA;
    float* deviceB;
    float* deviceC;

	GpuTimer timer0;
	GpuTimer timer1;
	GpuTimer timer2;
	timer0.Start();
    hipMalloc((void**)&deviceA, N * K * sizeof(float));
    hipMalloc((void**)&deviceB, K * M * sizeof(float));
    hipMalloc((void**)&deviceC, N * M * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(deviceA, hostA, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, K * M * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS initialization
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication
	timer0.Stop();
	int n_runs = 20;
	timer1.Start();
	for (int i = 0; i < n_runs; i++) {
    	CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, deviceA, N, deviceB, K, &beta, deviceC, N));
	}
	timer1.Stop();
	timer2.Start();

    // Copy output matrix from device to host
    hipMemcpy(hostC, deviceC, N * M * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    //for (int i = 0; i < N * M; ++i) {
    //    std::cout << hostC[i] << " ";
    //}
    //std::cout << std::endl;

    // Cleanup
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    delete[] hostA;
    delete[] hostB;
    delete[] hostC;

    hipblasDestroy(cublasHandle);
	timer2.Stop();
	std::cout << "Pre: " << timer0.Elapsed() << " Exec: " << timer1.Elapsed() / n_runs << " Post: " << timer2.Elapsed() << std::endl;


    return 0;
}
